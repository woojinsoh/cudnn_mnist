#include <hipDNN.h>
#include <hip/hip_runtime.h>
#include <iostream>

#include "model.h"
#include "utils.h"
#include "loss.h"


void Model::createHandles()
{
    checkCUDNN(hipdnnCreate(&cudnnHandle));        
    checkCUDNN(hipdnnCreateTensorDescriptor(&srcTensorDesc));
    checkCUDNN(hipdnnCreateTensorDescriptor(&dstTensorDesc));
    checkCUDNN(hipdnnCreateTensorDescriptor(&biasTensorDesc));
    checkCUDNN(hipdnnCreateFilterDescriptor(&filterDesc));
    checkCUDNN(hipdnnCreateConvolutionDescriptor(&convDesc));
    checkCUDNN(hipdnnCreatePoolingDescriptor(&poolingDesc));
    checkCUDNN(hipdnnCreateActivationDescriptor(&activationDesc));

    checkCUBLAS(hipblasCreate(&cublasHandle));
}   

void Model::destroyHandles()
{
    checkCUDNN(hipdnnDestroy(cudnnHandle));        
    checkCUDNN(hipdnnDestroyTensorDescriptor(srcTensorDesc));
    checkCUDNN(hipdnnDestroyTensorDescriptor(dstTensorDesc));
    checkCUDNN(hipdnnDestroyTensorDescriptor(biasTensorDesc));
    checkCUDNN(hipdnnDestroyFilterDescriptor(filterDesc));
    checkCUDNN(hipdnnDestroyConvolutionDescriptor(convDesc));
    checkCUDNN(hipdnnDestroyPoolingDescriptor(poolingDesc));
    checkCUDNN(hipdnnDestroyActivationDescriptor(activationDesc));    
}

Model::Model(){
    createHandles();
}

Model::~Model(){
    destroyHandles();
}

void Model::addLayers(Layer *layer)
{
    (*layer).setCudnnDescriptor(cudnnHandle, 
                               srcTensorDesc, 
                               dstTensorDesc, 
                               biasTensorDesc,
                               filterDesc, 
                               convDesc, 
                               poolingDesc, 
                               activationDesc);
                               
    (*layer).setCublasHandler(cublasHandle);
    model.push_back(layer);
}

ImageDto Model::Forward(ImageDto &data)
{
    ImageDto output = data;
    for(auto layer : model)
    {
        output = (*layer).Forward(output);
        checkCUDA(hipDeviceSynchronize());
    }
    return output;
}

ImageDto Model::Backward(ImageDto &data, int *labels)
{
    ImageDto output = data;
    for(auto layer = model.crbegin(); layer != model.crend(); layer++)
    {
        output = (*layer)->Backward(output, labels);//output = (*layer).Backward(output, labels);
        (*layer)->info_flag = 0;
    }
    return output;
}

void Model::Update(float learning_rate)
{
    for(auto layer : model)
    {
        (*layer).updateWeightBias(learning_rate);      
    }
}

float Model::Loss(ImageDto &data, int *onehot_labels_d, hipdnnSoftmaxAlgorithm_t softmax_algo)
{
    if(loss == nullptr) checkCUDA(hipMallocManaged(&loss, sizeof(float)));
    *loss = 0.f;
    
    if(softmax_algo==HIPDNN_SOFTMAX_LOG){
        nLLLoss<<<data.batch_size, data.num_features>>>(data.batch_size, data.buffer_d, onehot_labels_d, loss);
    }else{
        nLLoss<<<data.batch_size, data.num_features>>>(data.batch_size, data.buffer_d, onehot_labels_d, loss);
    }
    checkCUDA(hipDeviceSynchronize());
    
    return *loss;
}

float Model::Accuracy(ImageDto &data, int *labels_d, int num_classes)
{
    if(accuracy == nullptr) checkCUDA(hipMallocManaged(&accuracy, sizeof(float)));
    *accuracy = 0.f;

    calAccuracy<<<1, data.batch_size, data.batch_size>>>(data.batch_size, num_classes, data.buffer_d, labels_d, accuracy); // batchsize should be better for multiple of 32, and less than 2048.
    checkCUDA(hipDeviceSynchronize());

    return *accuracy;
}